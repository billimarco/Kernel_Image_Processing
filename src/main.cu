#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <random>
#include <ctime>
#include <iomanip>
#include <filesystem>
#include <stdexcept>
#include <sstream>
#include <optional>
#include <chrono>
#include <hip/hip_runtime.h>

#include <sys/stat.h>  // Per creare cartelle
#include <sys/types.h>
#ifdef _WIN32
    #include <direct.h>
    #define MKDIR(path) _mkdir(path)
#else
    #include <unistd.h>
    #define MKDIR(path) mkdir(path, 0777)
#endif

#define STB_IMAGE_IMPLEMENTATION
#include <stb/stb_image.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb/stb_image_write.h>

#include <nlohmann/json.hpp>
#include <Eigen/Dense>

using json = nlohmann::json;
const json CONFIG = json::parse(std::ifstream("settings/config.json"));
// change OMP_NUM_THREADS environment variable to run with 1 to X threads...
// check configuration in drop down menu
// XXX check working directory so that ./images and ./output are valid !

struct STBImage {
    int width{0}, height{0}, channels{0};
    uint8_t *image_data{nullptr};
    std::string filename{};

    // Funzione per caricare un'immagine
    bool loadImage(const std::string &name) {
        image_data = stbi_load(name.c_str(), &width, &height, &channels, 3); // Immagine rgb (3 canale) rimuovendo un eventuale canale Alpha
        if (channels == 4)
            channels = 3; //questo perchè se le immagini caricate sono RGBA ritorna il valore 4 su channels anche se quelli caricati sono solo 3
        if (!image_data)
            return false;
        else {
            filename = name;
            return true;
        }
    }

    // Funzione per salvare l'immagine
    void saveImage(const std::string &newName) const {
        stbi_write_jpg(newName.c_str(), width, height, channels, image_data, width);
    }

    // Funzione per inizializzare un'immagine RGB
    void initializeRGB(int w, int h) {
        width = w;
        height = h;
        channels = 3; // Immagine rgb con 3 canale
        image_data = (uint8_t*)malloc(width * height * channels);
    }
};

struct Kernel {
    std::vector<std::vector<float>> matrix;
    int size;

    // Costruttore
    Kernel(int s) : size(s) {
        matrix.resize(size, std::vector<float>(size, 0));
    }

    // Costruttore con inizializzazione e normalizzazione opzionale
    Kernel(int s, std::vector<std::vector<float>> initMatrix, bool normalizeKernel = true) : size(s) {
        if (initMatrix.size() != size || anyRowInvalid(initMatrix)) {
            throw std::invalid_argument("La matrice deve essere quadrata e della dimensione specificata.");
        }
        matrix = initMatrix;
        if (normalizeKernel) {
            normalize();
        }
    }

    // Metodo per verificare che tutte le righe abbiano la lunghezza corretta
    bool anyRowInvalid(const std::vector<std::vector<float>>& mat) {
        for (const auto& row : mat) {
            if (row.size() != size) {
                return true;
            }
        }
        return false;
    }

    // Metodo per normalizzare il kernel
    void normalize() {
        float sum = 0.0;
        for (const auto& row : matrix) {
            for (float value : row) {
                sum += value;
            }
        }
        if (sum != 0) {
            for (auto& row : matrix) {
                for (float& value : row) {
                    value /= sum;
                }
            }
        }
    }

    // Funzione per verificare se il kernel è separabile
    bool isSeparable() const {
        // Convertiamo la matrice in una matrice Eigen
        Eigen::MatrixXf mat(size, size);
        for (int i = 0; i < size; ++i) {
            for (int j = 0; j < size; ++j) {
                mat(i, j) = matrix[i][j];
            }
        }
    
        // Calcoliamo la decomposizione SVD della matrice
        Eigen::JacobiSVD<Eigen::MatrixXf> svd(mat, Eigen::ComputeThinU | Eigen::ComputeThinV);
    
        // Otteniamo i valori singolari
        Eigen::VectorXf singularValues = svd.singularValues();

        //std::cout << "SingularValues SVD (se uno solo è un valore non nullo è separabile):\n" << singularValues << std::endl;
    
        // Verifica se il numero di valori singolari non nulli è 1 (indica che la matrice ha rango 1)
        return singularValues(0) > 1e-6f && singularValues.tail(singularValues.size() - 1).isZero(1e-6f);
    }

    bool separate(std::vector<float>& vertical, std::vector<float>& horizontal) const {
        // Convertiamo la matrice in una matrice Eigen
        Eigen::MatrixXf mat(size, size);
        for (int i = 0; i < size; ++i) {
            for (int j = 0; j < size; ++j) {
                mat(i, j) = matrix[i][j];
            }
        }
    
        // Calcoliamo la decomposizione SVD della matrice
        Eigen::JacobiSVD<Eigen::MatrixXf> svd(mat, Eigen::ComputeThinU | Eigen::ComputeThinV);
    
        // Otteniamo i valori singolari
        Eigen::VectorXf singularValues = svd.singularValues();

        //std::cout << "SingularValues SVD (se uno solo è un valore non nullo è separabile):\n" << singularValues << std::endl;
    
        // Se il rango è maggiore di 1, il kernel non è separabile
        if (singularValues(0) < 1e-6f || !singularValues.tail(singularValues.size() - 1).isZero(1e-6f)) {
            return false;
        }
    
        // Otteniamo i vettori singolari
        Eigen::VectorXf u = svd.matrixU().col(0);  // Primo vettore di U (vettore colonna)
        Eigen::VectorXf v = svd.matrixV().col(0);  // Primo vettore di V (vettore riga)
    
        // Copiamo i vettori in std::vector
        vertical.assign(u.data(), u.data() + u.size());  // Vettore colonna in v
        horizontal.assign(vertical.data(), vertical.data() + vertical.size());  // Vettore riga in h

        //std::cout << "Vettore di convoluzione verticale:\n" << u << std::endl;
        //std::cout << "Vettore di convoluzione orizzontale:\n" << v << std::endl;
    
        return true;
    }

    // Metodo per stampare il kernel
    void print() const {
        for (const auto& row : matrix) {
            for (float value : row) {
                std::cout << value << " ";
            }
            std::cout << std::endl;
        }
    }
};

// Funzione per creare un cammino di cartelle
void createPath(const std::string &path) {
    std::istringstream ss(path);
    std::string partialPath;
    std::vector<std::string> directories;
    
    // Dividere il percorso nelle singole directory
    while (std::getline(ss, partialPath, '/')) {
        directories.push_back(partialPath);
    }

    std::string currentPath;
    for (const auto &dir : directories) {
        if (!currentPath.empty()) {
            currentPath += "/";
        }
        currentPath += dir;

        struct stat info;
        if (stat(currentPath.c_str(), &info) != 0) { // Se la cartella non esiste
            MKDIR(currentPath.c_str());
        }
    }
}

// Funzione per caricare immagini in un vettore
std::vector<STBImage> loadImages(const std::string& directory) {
    std::vector<STBImage> images;
    for (const auto& entry : std::filesystem::directory_iterator(directory)) {
        if (entry.is_regular_file()) {
            std::string filename = entry.path().string();
            STBImage img;
            if (img.loadImage(filename)) {
                images.push_back(img);
            }
        }
    }
    return images;
}

// Dummy CUDA kernel for warmup
__global__ void dummyKernel() {
}

void cudaWarmup() {
    // Launch a small kernel with 1 block and 1 thread (just to initialize GPU)
    auto start = std::chrono::high_resolution_clock::now();
    dummyKernel<<<1, 1>>>();
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration<double, std::milli>(end - start).count();
    
    // Wait for the kernel to finish
    hipDeviceSynchronize();
    
    // Check for any errors that might have occurred
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA warmup fallito: " << hipGetErrorString(err) << std::endl;
    } else {
        std::cout << "CUDA warmup successo:" << std::endl;
        std::cout << "Tempo di esecuzione CUDA warmup: " << duration << " ms" << std::endl;
    }
}
//<------------------------------------------------------------------------>
// Funzioni per la normale convoluzione RGB

// Funzione per fare Image Processing tramite convoluzione di un Kernel
STBImage convolveRGB(const STBImage &img, const Kernel &kernel) {
    int kCenter = kernel.size / 2;

    STBImage outputImg;
    outputImg.initializeRGB(img.width, img.height);

    for (int y = 0; y < img.height; y++) {
        for (int x = 0; x < img.width; x++) {
            for (int c = 0; c < img.channels; c++) {
                float sum = 0;
                for (int i = 0; i < kernel.size; i++) {
                    for (int j = 0; j < kernel.size; j++) {
                        int nx = x + j - kCenter;
                        int ny = y + i - kCenter;

                        // Zero-padding: Se fuori dai bordi, usa 0
                        if (nx < 0 || nx >= img.width || ny < 0 || ny >= img.height) {
                            sum += 0;
                        } else {
                            int pixelIndex = (ny * img.width + nx) * img.channels + c;
                            sum += img.image_data[pixelIndex] * kernel.matrix[i][j];
                        }
                    }
                }
                int newIndex = (y * img.width + x) * img.channels + c;
                outputImg.image_data[newIndex] = std::min(std::max(int(sum), 0), 255);
            }
        }
    }
    return outputImg;
}

// Funzione di convoluzione con un kernel in CUDA
__global__ void convolveKernelRGB(uint8_t *input, uint8_t *output, const float *kernel, int width, int height, int kernelSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int kCenter = kernelSize / 2;
    int pixelIndex = (y * width + x) * 3;
    float sum[3] = {0};

    if (x < width && y < height) return;

    for (int i = 0; i < kernelSize; i++) {
        for (int j = 0; j < kernelSize; j++) {
            int nx = x + j - kCenter;
            int ny = y + i - kCenter;
            
            if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                int idx = (ny * width + nx) * 3;
                float k = kernel[i * kernelSize + j];
                sum[0] += input[idx] * k;
                sum[1] += input[idx + 1] * k;
                sum[2] += input[idx + 2] * k;
            }
        }
    }

    output[pixelIndex] = min(max(int(sum[0]), 0), 255);
    output[pixelIndex + 1] = min(max(int(sum[1]), 0), 255);
    output[pixelIndex + 2] = min(max(int(sum[2]), 0), 255);
}

// Funzione per fare Image Processing tramite convoluzione di un Kernel con CUDA
STBImage convolveRGB_CUDA(const STBImage &img, const Kernel &kernel) {
    std::vector<float> h_allKernel(kernel.size * kernel.size);
    for (int i = 0; i < kernel.size; i++)
        for (int j = 0; j < kernel.size; j++)
            h_allKernel[i * kernel.size + j] = kernel.matrix[i][j];
    
    STBImage outputImg;
    outputImg.initializeRGB(img.width, img.height);
    
    int imageSize = img.width * img.height * 3;
    
    uint8_t *d_input, *d_output;
    float *d_kernel;

    hipMalloc(&d_input, imageSize);
    hipMalloc(&d_output, imageSize);
    hipMalloc(&d_kernel, kernel.size * kernel.size * sizeof(float));
    
    hipMemcpy(d_input, img.image_data, imageSize, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_allKernel.data(), kernel.size * kernel.size * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 blockSize(16, 16);
    dim3 gridSize((img.width + blockSize.x - 1) / blockSize.x, (img.height + blockSize.y - 1) / blockSize.y);
    
    convolveKernelRGB<<<gridSize, blockSize>>>(d_input, d_output, d_kernel, img.width, img.height, kernel.size);
    hipDeviceSynchronize();

    hipMemcpy(outputImg.image_data, d_output, imageSize, hipMemcpyDeviceToHost);
    
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);

    return outputImg;
}

//<------------------------------------------------------------------------>
// Funzioni per la convoluzione RGB con kernel separato

STBImage separableConvolutionRGB(const STBImage &img, Kernel &kernel) {
    std::vector<float> v, h;
    if (!kernel.separate(v, h)) {
        throw std::runtime_error("Il kernel non è separabile!");
    }

    int kCenter = kernel.size / 2;

    STBImage tempImg, outputImg;
    tempImg.initializeRGB(img.width, img.height);
    outputImg.initializeRGB(img.width, img.height);

    // Passo 1: Convoluzione orizzontale con zero-padding
    for (int y = 0; y < img.height; y++) {
        for (int x = 0; x < img.width; x++) {
            for (int c = 0; c < img.channels; c++) {
                float sum = 0;
                for (int k = 0; k < kernel.size; k++) {
                    int nx = x + k - kCenter;  // Calcola l'indice in base a k

                    // Zero padding: se l'indice è fuori dai limiti, usa zero
                    if (nx < 0 || nx >= img.width) {
                        sum += 0;  // Zero padding
                    } else {
                        int index = (y * img.width + nx) * img.channels + c;
                        sum += img.image_data[index] * h[k];
                    }
                }
                int tempIndex = (y * img.width + x) * img.channels + c;
                tempImg.image_data[tempIndex] = std::min(std::max(int(sum), 0), 255);
            }
        }
    }

    // Passo 2: Convoluzione verticale con zero-padding
    for (int y = 0; y < tempImg.height; y++) {
        for (int x = 0; x < tempImg.width; x++) {
            for (int c = 0; c < tempImg.channels; c++) {
                float sum = 0;
                for (int k = 0; k < kernel.size; k++) {
                    int ny = y + k - kCenter;  // Calcola l'indice in base a k

                    // Zero padding: se l'indice è fuori dai limiti, usa zero
                    if (ny < 0 || ny >= tempImg.height) {
                        sum += 0;  // Zero padding
                    } else {
                        int index = (ny * tempImg.width + x) * tempImg.channels + c;
                        sum += tempImg.image_data[index] * v[k];
                    }
                }
                int finalIndex = (y * tempImg.width + x) * tempImg.channels + c;
                outputImg.image_data[finalIndex] = std::min(std::max(int(sum), 0), 255);
            }
        }
    }

    delete[] tempImg.image_data;  // Libera la memoria temporanea
    return outputImg;
}

// CUDA Kernel for the horizontal convolution (first pass)
__global__ void convolutionHorizontalKernelRGB(const unsigned char *input, unsigned char *tempOutput, const float *hKernel, int width, int height, int kernelSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int kCenter = kernelSize / 2;
    int pixelIndex = (y * width + x) * 3;
    float sum[3] = {0};

    if (x < width && y < height) return;

    for (int k = 0; k < kernelSize; k++) {
        int nx = x + k - kCenter;
        if (nx >= 0 && nx < width) {
            int idx = (y * width + nx) * 3;

            sum[0] += input[idx] * hKernel[k];
            sum[1] += input[idx + 1] * hKernel[k];
            sum[2] += input[idx + 2] * hKernel[k];
        }
    }

    tempOutput[pixelIndex] = min(max(int(sum[0]), 0), 255);
    tempOutput[pixelIndex + 1] = min(max(int(sum[1]), 0), 255);
    tempOutput[pixelIndex + 2] = min(max(int(sum[2]), 0), 255);    
}

// CUDA Kernel for the vertical convolution (second pass)
__global__ void convolutionVerticalKernelRGB(const unsigned char *tempInput, unsigned char *output, const float *vKernel, int width, int height, int kernelSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int kCenter = kernelSize / 2;
    int pixelIndex = (y * width + x) * 3;
    float sum[3] = {0};

    if (x < width && y < height) return;

    
    for (int k = 0; k < kernelSize; k++) {
        int ny = y + k - kCenter;
        if (ny >= 0 && ny < height) {
            int idx = (ny * width + x) * 3;

            sum[0] += tempInput[idx] * vKernel[k];
            sum[1] += tempInput[idx + 1] * vKernel[k];
            sum[2] += tempInput[idx + 2] * vKernel[k];
        }
    }

    output[pixelIndex] = min(max(int(sum[0]), 0), 255);
    output[pixelIndex + 1] = min(max(int(sum[1]), 0), 255);
    output[pixelIndex + 2] = min(max(int(sum[2]), 0), 255); 
}

// Main CUDA function for separable convolution
STBImage separableConvolutionRGB_CUDA(const STBImage &img, Kernel &kernel) {
    std::vector<float> v, h;
    if (!kernel.separate(v, h)) {
        throw std::runtime_error("Il kernel non è separabile!");
    }

    STBImage tempImg, outputImg;
    tempImg.initializeRGB(img.width, img.height);
    outputImg.initializeRGB(img.width, img.height);

    int imageSize = img.width * img.height * 3;

    uint8_t *d_input, *d_tempOutput, *d_output;
    float *d_hKernel, *d_vKernel;

    hipMalloc(&d_input, imageSize);
    hipMalloc(&d_tempOutput, imageSize);
    hipMalloc(&d_output, imageSize);
    hipMalloc(&d_hKernel, kernel.size * sizeof(float));
    hipMalloc(&d_vKernel, kernel.size * sizeof(float));

    hipMemcpy(d_input, img.image_data, imageSize, hipMemcpyHostToDevice);
    hipMemcpy(d_hKernel, h.data(), kernel.size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vKernel, v.data(), kernel.size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((img.width + blockSize.x - 1) / blockSize.x, (img.height + blockSize.y - 1) / blockSize.y);

    convolutionHorizontalKernelRGB<<<gridSize, blockSize>>>(d_input, d_tempOutput, d_hKernel, img.width, img.height, kernel.size);
    hipDeviceSynchronize();

    convolutionVerticalKernelRGB<<<gridSize, blockSize>>>(d_tempOutput, d_output, d_vKernel, img.width, img.height, kernel.size);
    hipDeviceSynchronize();

    hipMemcpy(outputImg.image_data, d_output, img.width * img.height * img.channels * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_tempOutput);
    hipFree(d_output);
    hipFree(d_hKernel);
    hipFree(d_vKernel);

    return outputImg;
}
//<------------------------------------------------------------------------>
int main(){

    createPath("images/convolution");
    createPath("images/convolutionCUDA");
    createPath("images/separable_convolution");
    createPath("images/separable_convolutionCUDA");

    cudaWarmup();

    int num_images = CONFIG["num_images"];

    std::vector<STBImage> loadedImages = loadImages("images/basis");
    std::cout << "Totale immagini caricate: " << loadedImages.size() << std::endl;

    try {
        std::vector<std::vector<float>> sharpening = {{0, -1, 0}, 
                                                      {-1, 5, -1}, 
                                                      {0, -1, 0}};  // Esempio di Sharpen
        
        std::vector<std::vector<float>> gaussian_blur = {{1, 2, 1}, 
                                                         {2, 4, 2}, 
                                                         {1, 2, 1}}; // Esempio di Blur Gaussiano
        Kernel k(3, gaussian_blur, true);
        k.print();

        // Applica la convoluzione a tutte le immagini
        for (size_t i = 0; i < loadedImages.size(); i++) {

            auto start = std::chrono::high_resolution_clock::now();
            STBImage result1 = convolveRGB(loadedImages[i], k);
            auto end = std::chrono::high_resolution_clock::now();
            auto duration = std::chrono::duration<double, std::milli>(end - start).count();
            std::string originalFilename1 = std::filesystem::path(loadedImages[i].filename).filename().string();
            std::string outputFilename1 = "images/convolution/" + originalFilename1;
            result1.saveImage(outputFilename1);
            std::cout << "Immagine salvata come " << outputFilename1 << std::endl;
            std::cout << "Tempo di esecuzione: " << duration << " ms" << std::endl;

            auto start2 = std::chrono::high_resolution_clock::now();
            STBImage result2 = convolveRGB_CUDA(loadedImages[i], k);
            auto end2 = std::chrono::high_resolution_clock::now();
            auto duration2 = std::chrono::duration<double, std::milli>(end2 - start2).count();
            std::string originalFilename2 = std::filesystem::path(loadedImages[i].filename).filename().string();
            std::string outputFilename2 = "images/convolutionCUDA/" + originalFilename2;
            result1.saveImage(outputFilename2);
            std::cout << "Immagine salvata come " << outputFilename2 << std::endl;
            std::cout << "Tempo di esecuzione: " << duration2 << " ms" << std::endl;

            if(k.isSeparable()){
                auto start3 = std::chrono::high_resolution_clock::now();
                STBImage result3 = separableConvolutionRGB(loadedImages[i], k);
                auto end3 = std::chrono::high_resolution_clock::now();
                auto duration3 = std::chrono::duration<double, std::milli>(end3 - start3).count();
                std::string originalFilename3 = std::filesystem::path(loadedImages[i].filename).filename().string();
                std::string outputFilename3 = "images/separable_convolution/" + originalFilename3;
                result1.saveImage(outputFilename3);
                std::cout << "Immagine salvata come " << outputFilename3 << std::endl;
                std::cout << "Tempo di esecuzione: " << duration3 << " ms" << std::endl;

                auto start4 = std::chrono::high_resolution_clock::now();
                STBImage result4 = separableConvolutionRGB_CUDA(loadedImages[i], k);
                auto end4 = std::chrono::high_resolution_clock::now();
                auto duration4 = std::chrono::duration<double, std::milli>(end4 - start4).count();
                std::string originalFilename4 = std::filesystem::path(loadedImages[i].filename).filename().string();
                std::string outputFilename4 = "images/separable_convolutionCUDA/" + originalFilename4;
                result1.saveImage(outputFilename4);
                std::cout << "Immagine salvata come " << outputFilename4 << std::endl;
                std::cout << "Tempo di esecuzione: " << duration4 << " ms" << std::endl;
            }
        }
    } catch (const std::exception& e) {
        std::cerr << "Errore: " << e.what() << std::endl;
    }

    return 0;
}
